#include "hip/hip_runtime.h"
// followinng mvc_mesh.cpp

#include <hip/hip_vector_types.h>
#include <vector_functions.h>

#include <chrono>
#include <string>

#include "mesh_io.hpp"

constexpr float pi = 3.14159265358979323846;
constexpr float eps = 1e-6;

constexpr int BLOCK_SIZE = 256;

struct Face_Vertices
{
    float3 v0;
    float3 v1;
    float3 v2;
};

__device__ float3 operator+(const float3 &a, const float3 &b)
{
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__device__ float3 operator-(const float3 &a, const float3 &b)
{
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ float3 operator*(const float3 &a, const float b)
{
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ float3 operator*(const float a, const float3 &b)
{
    return make_float3(a * b.x, a * b.y, a * b.z);
}

__device__ float3 operator/(const float3 &a, const float b)
{
    return make_float3(a.x / b, a.y / b, a.z / b);
}

__device__ float dot(const float3 &a, const float3 &b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ float3 cross(const float3 &a, const float3 &b)
{
    return make_float3(a.y * b.z - a.z * b.y,
                       a.z * b.x - a.x * b.z,
                       a.x * b.y - a.y * b.x);
}

__global__ void mvc_mesh_kernel(const float3 *__restrict__ points,
                                float3 *__restrict__ result,
                                const Face_Vertices *__restrict__ from_vs,
                                const Face_Vertices *__restrict__ to_vs,
                                const int point_count,
                                const int face_count)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= point_count)
    {
        return;
    }
    const float3 point = points[idx];
    float total_w = 0;
    float3 total_f = {0.0f, 0.0f, 0.0f};
    for (int i = 0; i < face_count; i++)
    {
        const auto from_t1 = from_vs[i].v0;
        const auto from_t2 = from_vs[i].v1;
        const auto from_t3 = from_vs[i].v2;
        const auto to_t1 = to_vs[i].v0;
        const auto to_t2 = to_vs[i].v1;
        const auto to_t3 = to_vs[i].v2;
        const auto v1 = from_t1 - point;
        const auto v2 = from_t2 - point;
        const auto v3 = from_t3 - point;
        const auto u1 = v1 / norm3df(v1.x, v1.y, v1.z);
        const auto u2 = v2 / norm3df(v2.x, v2.y, v2.z);
        const auto u3 = v3 / norm3df(v3.x, v3.y, v3.z);
        const auto u23 = u2 - u3;
        const auto u31 = u3 - u1;
        const auto u12 = u1 - u2;
        const auto l1 = norm3df(u23.x, u23.y, u23.z);
        const auto l2 = norm3df(u31.x, u31.y, u31.z);
        const auto l3 = norm3df(u12.x, u12.y, u12.z);
        const auto theta1 = 2 * asinf(min(l1 / 2, 1.0));
        const auto theta2 = 2 * asinf(min(l2 / 2, 1.0));
        const auto theta3 = 2 * asinf(min(l3 / 2, 1.0));
        const auto h = (theta1 + theta2 + theta3) / 2;
        if (abs(pi - h) < eps)
        {
            const auto w1 = (tanf(theta2 / 2) + tanf(theta3 / 2)) / norm3df(v1.x, v1.y, v1.z);
            const auto w2 = (tanf(theta3 / 2) + tanf(theta1 / 2)) / norm3df(v2.x, v2.y, v2.z);
            const auto w3 = (tanf(theta1 / 2) + tanf(theta2 / 2)) / norm3df(v3.x, v3.y, v3.z);
            total_w = w1 + w2 + w3;
            total_f = w1 * to_t1 + w2 * to_t2 + w3 * to_t3;
            result[idx] = total_f / total_w;
            return;
        }
        else
        {
            auto n1 = cross(u2, u3);
            n1 = n1 * rnorm3df(n1.x, n1.y, n1.z);
            auto n2 = cross(u3, u1);
            n2 = n2 * rnorm3df(n2.x, n2.y, n2.z);
            auto n3 = cross(u1, u2);
            n3 = n3 * rnorm3df(n3.x, n3.y, n3.z);
            const auto det = dot(u1, cross(u2, u3));
            if (abs(det) < eps)
            {
                continue;
            }
            const auto m = (theta1 * n1 + theta2 * n2 + theta3 * n3) / 2;
            const auto w1 = dot(n1, m) / dot(n1, v1);
            const auto w2 = dot(n2, m) / dot(n2, v2);
            const auto w3 = dot(n3, m) / dot(n3, v3);
            total_w += w1 + w2 + w3;
            total_f = total_f + w1 * to_t1 + w2 * to_t2 + w3 * to_t3;
        }
    }
    result[idx] = total_f / total_w;
    return;
}

void test_mvc_mesh(const std::string &filename,
                   const std::string &cage_filename,
                   const std::string &deformed_cage_filename,
                   const std::string &output_filename)
{
    const auto start = std::chrono::high_resolution_clock::now();

    // read data/QMVC/FireHydrant.obj
    std::vector<std::array<float, 3>> vs;
    std::vector<std::vector<int>> fs;
    if (!read_any_obj(filename, vs, fs))
        return;
    // read cage test/FireHydrant_Cage_Triangulated.obj
    std::vector<std::array<float, 3>> from_vs;
    std::vector<std::array<int, 3>> from_fs;
    if (!read_tri_obj(cage_filename, from_vs, from_fs))
        return;
    // read defromed cage test/FireHydrant_Cage_Deformed_Triangulated.obj
    std::vector<std::array<float, 3>> to_vs;
    std::vector<std::array<int, 3>> to_fs;
    if (!read_tri_obj(deformed_cage_filename, to_vs, to_fs))
        return;

    const auto read = std::chrono::high_resolution_clock::now();

    const int vertex_count = vs.size();
    std::vector<float3> vs_f3(vertex_count);
    for (int i = 0; i < vertex_count; i++)
    {
        vs_f3[i] = make_float3(vs[i][0], vs[i][1], vs[i][2]);
    }
    const int face_count = from_fs.size();
    std::vector<Face_Vertices> from_face_vs(face_count);
    for (int i = 0; i < face_count; i++)
    {
        from_face_vs[i].v0 = make_float3(from_vs[from_fs[i][0]][0], from_vs[from_fs[i][0]][1], from_vs[from_fs[i][0]][2]);
        from_face_vs[i].v1 = make_float3(from_vs[from_fs[i][1]][0], from_vs[from_fs[i][1]][1], from_vs[from_fs[i][1]][2]);
        from_face_vs[i].v2 = make_float3(from_vs[from_fs[i][2]][0], from_vs[from_fs[i][2]][1], from_vs[from_fs[i][2]][2]);
    }
    std::vector<Face_Vertices> to_face_vs(face_count);
    for (int i = 0; i < face_count; i++)
    {
        to_face_vs[i].v0 = make_float3(to_vs[to_fs[i][0]][0], to_vs[to_fs[i][0]][1], to_vs[to_fs[i][0]][2]);
        to_face_vs[i].v1 = make_float3(to_vs[to_fs[i][1]][0], to_vs[to_fs[i][1]][1], to_vs[to_fs[i][1]][2]);
        to_face_vs[i].v2 = make_float3(to_vs[to_fs[i][2]][0], to_vs[to_fs[i][2]][1], to_vs[to_fs[i][2]][2]);
    }

    hipDeviceSynchronize();
    const auto prepare = std::chrono::high_resolution_clock::now();

    float3 *d_vs;
    hipMalloc(&d_vs, vertex_count * sizeof(float3));
    hipMemcpy(d_vs, vs_f3.data(), vertex_count * sizeof(float3), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    const auto copy_vs = std::chrono::high_resolution_clock::now();

    Face_Vertices *d_from_face_vs;
    hipMalloc(&d_from_face_vs, face_count * sizeof(Face_Vertices));
    hipMemcpy(d_from_face_vs, from_face_vs.data(), face_count * sizeof(Face_Vertices), hipMemcpyHostToDevice);

    Face_Vertices *d_to_face_vs;
    hipMalloc(&d_to_face_vs, face_count * sizeof(Face_Vertices));
    hipMemcpy(d_to_face_vs, to_face_vs.data(), face_count * sizeof(Face_Vertices), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    const auto copy_face_vs = std::chrono::high_resolution_clock::now();

    float3 *d_result;
    hipMalloc(&d_result, vertex_count * sizeof(float3));
    mvc_mesh_kernel<<<(vertex_count + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(d_vs, d_result, d_from_face_vs, d_to_face_vs, vertex_count, face_count);

    hipDeviceSynchronize();
    const auto kernel = std::chrono::high_resolution_clock::now();

    std::vector<float3> result(vertex_count);
    hipMemcpy(result.data(), d_result, vertex_count * sizeof(float3), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    const auto write = std::chrono::high_resolution_clock::now();

    std::vector<std::array<float, 3>> result_vs(vertex_count);
    for (int i = 0; i < vertex_count; i++)
    {
        result_vs[i] = {float(result[i].x), float(result[i].y), float(result[i].z)};
    }
    if (!write_any_obj(output_filename, result_vs, fs))
        return;

    const auto end = std::chrono::high_resolution_clock::now();

    const auto read_duration = std::chrono::duration_cast<std::chrono::microseconds>(read - start).count();
    const auto prepare_duration = std::chrono::duration_cast<std::chrono::microseconds>(prepare - read).count();
    const auto copy_vs_duration = std::chrono::duration_cast<std::chrono::microseconds>(copy_vs - prepare).count();
    const auto copy_face_vs_duration = std::chrono::duration_cast<std::chrono::microseconds>(copy_face_vs - copy_vs).count();
    const auto kernel_duration = std::chrono::duration_cast<std::chrono::microseconds>(kernel - copy_face_vs).count();
    const auto copy_result_duration = std::chrono::duration_cast<std::chrono::microseconds>(write - kernel).count();
    const auto write_duration = std::chrono::duration_cast<std::chrono::microseconds>(end - write).count();

    std::cout << "read duration: " << read_duration << " us" << std::endl;
    std::cout << "prepare duration: " << prepare_duration << " us" << std::endl;
    std::cout << "copy vs duration: " << copy_vs_duration << " us" << std::endl;
    std::cout << "copy face vs duration: " << copy_face_vs_duration << " us" << std::endl;
    std::cout << "kernel duration: " << kernel_duration << " us" << std::endl;
    std::cout << "copy result duration: " << copy_result_duration << " us" << std::endl;
    std::cout << "write duration: " << write_duration << " us" << std::endl;

    hipFree(d_vs);
    hipFree(d_result);
    hipFree(d_from_face_vs);
    hipFree(d_to_face_vs);
}

int main(int argc, char **argv)
{
    if (argc != 5)
    {
        std::cerr << "Usage: " << argv[0] << " <filename> <cage_filename> <deformed_cage_filename> <output_filename>" << std::endl;
        return -1;
    }
    const std::string filename = argv[1];
    const std::string cage_filename = argv[2];
    const std::string deformed_cage_filename = argv[3];
    const std::string output_filename = argv[4];

    test_mvc_mesh(filename, cage_filename, deformed_cage_filename, output_filename);
    return 0;
}