#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <unordered_map>
#include <fstream>
#include <chrono>

#include "mesh_io.hpp"

using Point = std::array<float, 3>;

struct Face_Vertices
{
    float3 v0;
    float3 v1;
    float3 v2;
};

struct Mesh_By_vs_fs
{
    std::vector<Point> vs;
    std::vector<std::array<int, 3>> fs;
};

struct Mesh_and_Weights
{
    Mesh_By_vs_fs mesh;
    std::vector<std::vector<float>> all_weights;
};

const std::string ngon_prefix = "./ngon/";

int load_ngons(const std::vector<std::vector<int>> &fs,
               std::unordered_map<int, Mesh_and_Weights> &ngons)
{
    int total_face_count = 0;
    for (const auto f : fs)
    {
        const auto vertex_count = f.size();
        if (vertex_count < 3)
        {
            std::cerr << "Invalid face with less than 3 vertices:" << f[0] << " " << f[1] << " " << f[2] << std::endl;
            return -1;
        }
        if (ngons.find(vertex_count) == ngons.end())
        {
            const auto ngon_off_path = ngon_prefix + std::to_string(vertex_count) + "gon.off";
            const auto ngon_coord_path = ngon_prefix + std::to_string(vertex_count) + "gon.coord";
            Mesh_By_vs_fs mesh;
            read_tri_off(ngon_off_path, mesh.vs, mesh.fs);
            std::vector<std::vector<float>> all_weights;
            std::ifstream coord_file(ngon_coord_path);
            for (int i = 0; i < mesh.vs.size(); i++)
            {
                std::vector<float> weights;
                for (int j = 0; j < vertex_count; j++)
                {
                    float weight;
                    coord_file >> weight;
                    weights.push_back(weight);
                }
                all_weights.push_back(weights);
            }
            ngons[vertex_count] = {mesh, all_weights};
        }
        total_face_count += ngons.at(vertex_count).mesh.fs.size();
    }
    return total_face_count;
}

void triangulate_mesh(const std::vector<Point> &vs,
                      const std::vector<std::vector<int>> &fs,
                      std::vector<Face_Vertices> &result)
{
    std::unordered_map<int, Mesh_and_Weights> ngons;
    const auto total_face_count = load_ngons(fs, ngons);
    if (total_face_count < 0)
    {
        std::cerr << "Failed to load ngons" << std::endl;
        return;
    }
    result.clear();
    result.reserve(total_face_count);
    for (const auto f : fs)
    {
        const auto vertex_count = f.size();
        if (vertex_count < 3)
        {
            std::cerr << "Invalid face with less than 3 vertices" << std::endl;
            return;
        }
        {
            const auto &mesh = ngons.at(vertex_count).mesh;
            const auto &all_weights = ngons.at(vertex_count).all_weights;
            std::vector<Point> f_vertices;
            for (const auto v_index : f)
            {
                f_vertices.push_back(vs[v_index]);
            }
            std::vector<Point> new_vs(mesh.vs.size());
            for (int i = 0; i < mesh.vs.size(); i++)
            {
                Point new_v = {0, 0, 0};
                for (int j = 0; j < vertex_count; j++)
                {
                    new_v[0] += all_weights[i][j] * f_vertices[j][0];
                    new_v[1] += all_weights[i][j] * f_vertices[j][1];
                    new_v[2] += all_weights[i][j] * f_vertices[j][2];
                }
                new_vs[i] = new_v;
            }
            for (const auto &face : mesh.fs)
            {
                Face_Vertices face_vertices = {
                    make_float3(new_vs[face[0]][0], new_vs[face[0]][1], new_vs[face[0]][2]),
                    make_float3(new_vs[face[1]][0], new_vs[face[1]][1], new_vs[face[1]][2]),
                    make_float3(new_vs[face[2]][0], new_vs[face[2]][1], new_vs[face[2]][2]),
                };
                result.push_back(face_vertices);
            }
        }
    }
}

int main(int argc, char **argv)
{
    if (argc != 3)
    {
        std::cerr << "Usage: " << argv[0] << " input_filename output_filename" << std::endl;
        return -1;
    }
    const std::string input_path = argv[1];
    const std::string output_path = argv[2];

    const auto start = std::chrono::high_resolution_clock::now();

    std::vector<Point> vs;
    std::vector<std::vector<int>> fs;
    if (!read_any_obj(input_path, vs, fs))
        return -1;
    const auto read = std::chrono::high_resolution_clock::now();

    std::vector<Face_Vertices> result;
    triangulate_mesh(vs, fs, result);
    const auto triangulation = std::chrono::high_resolution_clock::now();

    const auto face_count = result.size();
    const auto vertex_count = face_count * 3;
    std::vector<Point> new_vs;
    new_vs.reserve(vertex_count);
    std::vector<std::array<int, 3>> new_fs;
    new_fs.reserve(face_count);
    int index = 0;
    for (const auto &face_vertices : result)
    {
        new_fs.push_back({index, index + 1, index + 2});
        new_vs.push_back({face_vertices.v0.x, face_vertices.v0.y, face_vertices.v0.z});
        new_vs.push_back({face_vertices.v1.x, face_vertices.v1.y, face_vertices.v1.z});
        new_vs.push_back({face_vertices.v2.x, face_vertices.v2.y, face_vertices.v2.z});
        index += 3;
    }
    write_tri_obj(output_path, new_vs, new_fs);
    const auto write = std::chrono::high_resolution_clock::now();

    const auto read_duration = std::chrono::duration_cast<std::chrono::microseconds>(read - start).count();
    const auto triangulation_duration = std::chrono::duration_cast<std::chrono::microseconds>(triangulation - read).count();
    const auto write_duration = std::chrono::duration_cast<std::chrono::microseconds>(write - triangulation).count();
    const auto total_duration = std::chrono::duration_cast<std::chrono::microseconds>(write - start).count();
    std::cout << "Read: " << read_duration << "us" << std::endl;
    std::cout << "Triangulation: " << triangulation_duration << "us" << std::endl;
    std::cout << "Write: " << write_duration << "us" << std::endl;
    std::cout << "Total: " << total_duration << "us" << std::endl;
    return 0;
}